#include "hip/hip_runtime.h"
/*
  cuda.cu - Image manipulations using CUDA
  Copyright (c) 2017 Elsa Buchholz, Florian Schwab
*/

#include <stdio.h>
#include <stdlib.h>
#include <opencv2/opencv.hpp>
#include "common.h"


#pragma mark Transformation Types

int const CUDA_SWAP   = 0;
int const CUDA_GRAY   = 1;
int const CUDA_BLUR   = 2;
int const CUDA_EMBOSS = 3;


#pragma mark Macros

// Length of an array
#define ARRAY_LENGTH(a) ((sizeof(a) > 0) ? sizeof(a) / sizeof(a[0]) : 0)

// Check if CUDA function was executed successfully
#define CUDA_CHECK(call) {                                                                                \
  const hipError_t e = call;                                                                             \
  if (e != hipSuccess) {                                                                                 \
    printf("\nCUDA error: %s:%d, code: %d, reason: %s\n", __FILE__, __LINE__, e, hipGetErrorString(e));  \
    exit(2);                                                                                              \
  }                                                                                                       \
}


#pragma mark Kernels

__global__ void kernel_swap(uint32_t *in, uint32_t *out, uint32_t w, uint32_t h) {
  int idx = blockIdx.y * w + blockIdx.x;

  // Check if thread index is no longer within input array
  if (ARRAY_LENGTH(in) >= idx) { return; }

  out[idx] = RGBA(RED(in[idx]), BLUE(in[idx]), GREEN(in[idx]), ALPHA(in[idx]));
}

__global__ void kernel_gray(uint32_t *in, uint32_t *out, uint32_t w, uint32_t h) {
  int idx = blockIdx.y * w + blockIdx.x;

  // Check if thread index is no longer within input array
  if (ARRAY_LENGTH(in) >= idx) { return; }

  uint8_t gray = (0.21 * RED(in[idx])) + (0.72 * GREEN(in[idx])) + (0.07 * BLUE(in[idx]));

  out[idx] = RGBA(gray, gray, gray, ALPHA(in[idx]));
}

__global__ void kernel_blur(uint32_t *in, uint32_t *out, uint32_t w, uint32_t h, uint8_t area) {
  int idx = blockIdx.y * w + blockIdx.x;

  // Check if thread index is no longer within input array
  if (ARRAY_LENGTH(in) >= idx) { return; }

  uint32_t min_x      = blockIdx.x < area ? 0 : blockIdx.x - area;
  uint32_t min_y      = blockIdx.y < area ? 0 : blockIdx.y - area;
  uint32_t max_x      = (blockIdx.x + area) >= w ? w : blockIdx.x + area;
  uint32_t max_y      = (blockIdx.y + area) >= h ? h : blockIdx.y + area;
  uint32_t num_pixels = 0;
  uint32_t red_sum    = 0;
  uint32_t green_sum  = 0;
  uint32_t blue_sum   = 0;
  uint32_t alpha_sum  = 0;
  uint32_t i          = 0;

  for(int x = min_x; x < max_x; x += 1) {
    for(int y = min_y; y < max_y; y += 1) {
      i = y * w + x;

      num_pixels += 1;
      red_sum    += RED(in[i]);
      green_sum  += GREEN(in[i]);
      blue_sum   += BLUE(in[i]);
      alpha_sum  += ALPHA(in[i]);
    }
  }

  out[idx] = RGBA((red_sum / num_pixels), (green_sum / num_pixels), (blue_sum / num_pixels), (alpha_sum / num_pixels));
}

__global__ void kernel_emboss(uint32_t *in, uint32_t *out, uint32_t w, uint32_t h) {
  if (blockIdx.y < 1 || blockIdx.x < 1) { return; }

  int idx     = blockIdx.y * w + blockIdx.x;
  int idx_ref = (blockIdx.y - 1) * w + (blockIdx.x - 1);

  // Check if thread index is no longer within input array
  if (ARRAY_LENGTH(in) >= idx) { return; }
  if (ARRAY_LENGTH(in) >= idx_ref) { return; }

  int diffs[] = {
    (RED(in[idx_ref]) - RED(in[idx])),
    (GREEN(in[idx_ref]) - GREEN(in[idx])),
    (BLUE(in[idx_ref]) - BLUE(in[idx]))
  };

  int diff = diffs[0];
  if ((diffs[1] < 0 ? diffs[1] * -1 : diffs[1]) > diff) { diff = diffs[1]; }
  if ((diffs[2] < 0 ? diffs[2] * -1 : diffs[2]) > diff) { diff = diffs[2]; }

  int gray = 128 + diff;
  if (gray > 255) { gray = 255; }
  if (gray < 0) { gray = 0; }

  out[idx] = RGBA(gray, gray, gray, ALPHA(in[idx]));
}


#pragma mark CUDA wrapper

// Output CUDA information
static void showCudaInfo() {
  hipDeviceProp_t prop;
  CUDA_CHECK(hipGetDeviceProperties(&prop, 0));

  printf("CUDA INFORMATION\n================\n");
  printf("Name: %s\n", prop.name);
  printf("Total Memory: %u Bytes\n", prop.totalGlobalMem);
  printf("Max. Threads Per Block: %d\n", prop.maxThreadsPerBlock);
  printf("Clock Rate: %d kHz\n", prop.clockRate);
  printf("Multiprocessors: %d\n", prop.multiProcessorCount);
  printf("Concurrent Kernels: %d\n", prop.concurrentKernels);
}

// Wrapper for all CUDA kernels
result cuda(int type, uint32_t width, uint32_t height, uint32_t *data, uint8_t area) {
  // Show CUDA infos
  #ifndef GNUPLOT_MODE
  showCudaInfo();
  #endif

  size_t buffer_size = width * height * sizeof(uint32_t);
  uint32_t *dev_in, *dev_out;

  // Allocate memory on device
  CUDA_CHECK(hipMalloc((void **) &dev_in, buffer_size));
  CUDA_CHECK(hipMalloc((void **) &dev_out, buffer_size));

  // Copy image data to device
  CUDA_CHECK(hipMemcpy(dev_in, data, buffer_size, hipMemcpyHostToDevice));

  // Measure execution time
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  dim3 threads(8, 8);
  dim3 blocks((width / threads.x + 1), (height / threads.y + 1));

  hipEventRecord(start);

  switch(type) {
    case CUDA_SWAP:
      kernel_swap<<<blocks, threads>>>(dev_in, dev_out, width, height);
      break;

    case CUDA_GRAY:
      kernel_gray<<<blocks, threads>>>(dev_in, dev_out, width, height);
      break;

    case CUDA_BLUR:
      kernel_blur<<<blocks, threads>>>(dev_in, dev_out, width, height, area);
      break;

    case CUDA_EMBOSS:
      kernel_emboss<<<blocks, threads>>>(dev_in, dev_out, width, height);
      break;
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float runtime = 0;
  hipEventElapsedTime(&runtime, start, stop);

  // Copy transformed image data from device
  CUDA_CHECK(hipMemcpy(data, dev_out, buffer_size, hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(dev_in));
  CUDA_CHECK(hipFree(dev_out));

  // Terminate CUDA device usage
  CUDA_CHECK(hipDeviceReset());

  struct result res;
  res.code    = RES_ARRAY;
  res.runtime = (long) (runtime * 1000);

  return res;
}


#pragma mark Transformations

result swap(cv::Mat *image, uint32_t width, uint32_t height, uint32_t *data) {
  return cuda(CUDA_SWAP, width, height, data, 0);
}

result gray(cv::Mat *image, uint32_t width, uint32_t height, uint32_t *data) {
  return cuda(CUDA_GRAY, width, height, data, 0);
}

result blur(cv::Mat *image, uint32_t width, uint32_t height, uint32_t *data, uint8_t area) {
  return cuda(CUDA_BLUR, width, height, data, area);
}

result emboss(cv::Mat *image, uint32_t width, uint32_t height, uint32_t *data) {
  return cuda(CUDA_EMBOSS, width, height, data, 0);
}
