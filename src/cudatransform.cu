#include "hip/hip_runtime.h"
/*
  cudatransform.cu - Image manipulations using CUDA
  Copyright (c) 2017 Elsa Buchholz, Florian Schwab
*/


#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <stdint.h>
#include "pnglite.h"


extern const char *__progname;


#pragma mark Image Helper Stuff

// Representation image
struct Image {
  png_t    png;
  uint32_t *pixels;
  size_t   number_of_pixels;
  uint     width;
  uint     height;
};

// Access red, green, blue, and alpha component values in a 32-bit unsigned RGBA pixel value.
#define ALPHA(pixel) ((pixel)>>24)
#define BLUE(pixel)  (((pixel)>>16)&0xFF)
#define GREEN(pixel) (((pixel)>>8)&0xFF)
#define RED(pixel)   ((pixel)&0xFF)

// Encode a 32-bit unsigned RGBA value from individual red, green, blue, and alpha component values.
#define RGBA(r,g,b,a) ((((a) << 24)) | (((b) << 16)) | (((g) << 8)) | ((r)))

// Length of an array
#define ARRAY_LENGTH(a) ((sizeof(a) > 0) ? sizeof(a) / sizeof(a[0]) : 0)

// Check if CUDA function was executed successfully
#define CUDA_CHECK(call) {                                                                                \
  const hipError_t e = call;                                                                             \
  if (e != hipSuccess) {                                                                                 \
    printf("\nCUDA error: %s:%d, code: %d, reason: %s\n", __FILE__, __LINE__, e, hipGetErrorString(e));  \
    exit(2);                                                                                              \
  }                                                                                                       \
}


#pragma mark CUDA Kernels

// Swap green and blue
__global__ void kernel_swap_green_blue(uint32_t *in, uint32_t *out, int w, int h){
  int idx = blockIdx.y * w + blockIdx.x;

  // Check if thread index is no longer within input array
  if (ARRAY_LENGTH(in) >= idx) { return; }

  out[idx] = RGBA(RED(in[idx]), BLUE(in[idx]), GREEN(in[idx]), ALPHA(in[idx]));
}

// Transform image into gray scale
__global__ void kernel_gray(uint32_t *in, uint32_t *out, int w, int h){
  int idx = blockIdx.y * w + blockIdx.x;

  // Check if thread index is no longer within input array
  if (ARRAY_LENGTH(in) >= idx) { return; }

  uint8_t gray = (0.21 * RED(in[idx])) + (0.72 * GREEN(in[idx])) + (0.07 * BLUE(in[idx]));

  out[idx] = RGBA(gray, gray, gray, ALPHA(in[idx]));
}

// Blur image
__global__ void kernel_blur(uint32_t *in, uint32_t *out, int w, int h, int area) {
  int idx = blockIdx.y * w + blockIdx.x;

  // Check if thread index is no longer within input array
  if (ARRAY_LENGTH(in) >= idx) { return; }

  uint32_t min_x      = blockIdx.x < area ? 0 : blockIdx.x - area;
  uint32_t min_y      = blockIdx.y < area ? 0 : blockIdx.y - area;
  uint32_t max_x      = (blockIdx.x + area) >= w ? w : blockIdx.x + area;
  uint32_t max_y      = (blockIdx.y + area) >= h ? h : blockIdx.y + area;
  uint32_t num_pixels = 0;
  uint32_t red_sum    = 0;
  uint32_t green_sum  = 0;
  uint32_t blue_sum   = 0;
  uint32_t alpha_sum  = 0;
  int      i          = 0;

  for(int x = min_x; x < max_x; x += 1) {
    for(int y = min_y; y < max_y; y += 1) {
      i = y * w + x;

      num_pixels += 1;
      red_sum    += RED(in[i]);
      green_sum  += GREEN(in[i]);
      blue_sum   += BLUE(in[i]);
      alpha_sum  += ALPHA(in[i]);
    }
  }

  out[idx] = RGBA((red_sum / num_pixels), (green_sum / num_pixels), (blue_sum / num_pixels), (alpha_sum / num_pixels));
}

// Transform image with emboss
__global__ void kernel_emboss(uint32_t *in, uint32_t *out, int w, int h) {
  if (blockIdx.y < 1 || blockIdx.x < 1) { return; }

  int idx     = blockIdx.y * w + blockIdx.x;
  int idx_ref = (blockIdx.y - 1) * w + (blockIdx.x - 1);

  // Check if thread index is no longer within input array
  if (ARRAY_LENGTH(in) >= idx) { return; }
  if (ARRAY_LENGTH(in) >= idx_ref) { return; }

  int diffs[] = {
    (RED(in[idx_ref]) - RED(in[idx])),
    (GREEN(in[idx_ref]) - GREEN(in[idx])),
    (BLUE(in[idx_ref]) - BLUE(in[idx]))
  };

  int diff = diffs[0];
  if ((diffs[1] < 0 ? diffs[1] * -1 : diffs[1]) > diff) { diff = diffs[1]; }
  if ((diffs[2] < 0 ? diffs[2] * -1 : diffs[2]) > diff) { diff = diffs[2]; }

  int gray = 128 + diff;
  if (gray > 255) { gray = 255; }
  if (gray < 0) { gray = 0; }

  out[idx] = RGBA(gray, gray, gray, ALPHA(in[idx]));
}


#pragma mark Helper Methods

// Terminate program with message
void terminate(const char *fmt, ...) {
  va_list args;

  va_start(args, fmt);
  vfprintf(stderr, fmt, args);
  va_end(args);

  exit(1);
}

// Allocate a buffer large enough to store pixel data for given image.
uint32_t *alloc_image_buffer(Image *img) {
  return (uint32_t *) malloc(img->number_of_pixels * sizeof(uint32_t));
}

// Read an image from a file
static Image *read_image(const char *filename) {
  Image *img = (Image *) malloc(sizeof(Image));

  if (png_open_file_read(&img->png, filename) != PNG_NO_ERROR) {
    terminate("Couldn't open image\n");
  }

  // Number of pixels
  img->width            = img->png.width;
  img->height           = img->png.height;
  img->number_of_pixels = img->png.width * img->png.height;

  if (img->png.color_type != PNG_TRUECOLOR_ALPHA) {
    terminate("Only true color alpha images supported\n");
  }

  img->pixels = alloc_image_buffer(img);

  if (png_get_data(&img->png, (unsigned char *) img->pixels) != PNG_NO_ERROR) {
    terminate("Could not read image data\n");
  }

  return img;
}

// Save a transformed image.
static void save_image(const char *filename, uint32_t *img_data, const Image *orig_img) {
  png_t out;

  if (png_open_file_write(&out, filename) != PNG_NO_ERROR) {
    terminate("couldn't open image to save\n");
  }

  if (png_set_data(&out, orig_img->png.width, orig_img->png.height, orig_img->png.depth,
                   orig_img->png.color_type, (unsigned char *)img_data) != PNG_NO_ERROR) {
    terminate("Could not save image data\n");
  }

  png_close_file(&out);
}


#pragma mark Info Outputs

// Output CUDA information
static void showCudaInfo() {
  hipDeviceProp_t prop;
  CUDA_CHECK(hipGetDeviceProperties(&prop, 0));

  printf("CUDA INFORMATION\n================\n");
  printf("Name: %s\n", prop.name);
  printf("Total Memory: %u Bytes\n", prop.totalGlobalMem);
  printf("Max. Threads Per Block: %d\n", prop.maxThreadsPerBlock);
  printf("Clock Rate: %d kHz\n", prop.clockRate);
  printf("Multiprocessors: %d\n", prop.multiProcessorCount);
  printf("Concurrent Kernels: %d\n", prop.concurrentKernels);
}

// Output image information
static void showImageInfo(const Image *img) {
  printf("\nIMAGE INFORMATION\n================\n");
  printf("Width: %u\n", img->width);
  printf("Height: %u\n", img->height);
  printf("Total Pixels: %u\n", img->number_of_pixels);
}


#pragma mark main

int main(int argc, char **argv) {
  if (argc < 4) {
    terminate("Usage: %s <swap|gray|blur|emboss> <infile> <outfile> (<area>)\n", __progname);
  }

  // Load image
  png_init(0, 0);

  Image    *img          = read_image(argv[2]);
  uint32_t *img_data     = img->pixels;
  uint32_t *out_img_data = (uint32_t *) alloc_image_buffer(img);

  // Show CUDA infos
  showCudaInfo();

  // Show image infos
  showImageInfo(img);

  // Initialize/allocate buffers
  size_t buffer_size = img->number_of_pixels * sizeof(uint32_t);
  uint32_t *dev_imgdata, *dev_imgdata_out;
  CUDA_CHECK(hipMalloc((void **) &dev_imgdata, buffer_size));
  CUDA_CHECK(hipMalloc((void **) &dev_imgdata_out, buffer_size));

  // Copy image data to device
  CUDA_CHECK(hipMemcpy(dev_imgdata, img_data, buffer_size, hipMemcpyHostToDevice));
  dim3 grid(img->width, img->height);

  // Switch transformation type
  if (strcmp(argv[1], "swap") == 0) {
    kernel_swap_green_blue<<<grid, 1>>>(dev_imgdata, dev_imgdata_out, img->width, img->height);
  } else if (strcmp(argv[1], "gray") == 0) {
    kernel_gray<<<grid, 1>>>(dev_imgdata, dev_imgdata_out, img->width, img->height);
  } else if (strcmp(argv[1], "blur") == 0) {
    int area = 11;

    if (argc == 5) { area = atoi(argv[4]); }

    kernel_blur<<<grid, 1>>>(dev_imgdata, dev_imgdata_out, img->width, img->height, area);
  } else if (strcmp(argv[1], "emboss") == 0) {
    kernel_emboss<<<grid, 1>>>(dev_imgdata, dev_imgdata_out, img->width, img->height);
  } else {
    terminate("\nUnsupported Transformation: %s\n", argv[1]);
  }

  // Copy transformed image data from device
  CUDA_CHECK(hipMemcpy(out_img_data, dev_imgdata_out, buffer_size, hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(dev_imgdata));
  CUDA_CHECK(hipFree(dev_imgdata_out));

  // Terminate CUDA device usage
  CUDA_CHECK(hipDeviceReset());

  // Save image to disk and close file handle
  save_image(argv[3], out_img_data, img);
  png_close_file(&img->png);

  printf("\nSaved Transformed Image: %s\n", argv[3]);

  // Cleanup memory
  free(img->pixels);
  free(img);
  free(out_img_data);

  return 0;
}
