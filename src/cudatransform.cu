#include "hip/hip_runtime.h"
/*
  cudatransform.cu - Image manipulations using CUDA
  Copyright (c) 2017 Elsa Buchholz, Florian Schwab
*/


#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <stdint.h>
#include "pnglite.h"


extern const char *__progname;


#pragma mark Image Helper Stuff

// Representation image
struct Image {
  png_t    png;
  uint32_t *pixels;
  size_t   number_of_pixels;
  uint     width;
  uint     height;
};

// Access red, green, blue, and alpha component values in a 32-bit unsigned RGBA pixel value.
#define ALPHA(pixel) ((pixel)>>24)
#define BLUE(pixel)  (((pixel)>>16)&0xFF)
#define GREEN(pixel) (((pixel)>>8)&0xFF)
#define RED(pixel)   ((pixel)&0xFF)

// Encode a 32-bit unsigned RGBA value from individual red, green, blue, and alpha component values.
#define RGBA(r,g,b,a) ((((a) << 24)) | (((b) << 16)) | (((g) << 8)) | ((r)))


#pragma mark CUDA Kernels

// Swap green and blue
__global__ void kernel_swap_green_blue(uint32_t *in, uint32_t *out, int w, int h){
  int idx = blockIdx.y * w + blockIdx.x;

  out[idx] = RGBA(RED(in[idx]), BLUE(in[idx]), GREEN(in[idx]), ALPHA(in[idx]));
}

// Transform image into gray scale
__global__ void kernel_gray(uint32_t *in, uint32_t *out, int w, int h){
  int idx = blockIdx.y * w + blockIdx.x;

  uint8_t gray = (0.21 * RED(in[idx])) + (0.72 * GREEN(in[idx])) + (0.07 * BLUE(in[idx]));

  out[idx] = RGBA(gray, gray, gray, ALPHA(in[idx]));
}

// Blur image
__global__ void kernel_blur(uint32_t *in, uint32_t *out, int w, int h) {
  int idx = blockIdx.y * w + blockIdx.x;

  // TODO implement

  out[idx] = RGBA(RED(in[idx]), GREEN(in[idx]), BLUE(in[idx]), ALPHA(in[idx]));
}

// Transform image with emboss
__global__ void kernel_emboss(uint32_t *in, uint32_t *out, int w, int h) {
  if (blockIdx.y < 1 || blockIdx.x < 1) { return; }

  int idx     = blockIdx.y * w + blockIdx.x;
  int idx_ref = (blockIdx.y - 1) * w + (blockIdx.x - 1);

  int diffs[] = {
    (RED(in[idx_ref]) - RED(in[idx])),
    (GREEN(in[idx_ref]) - GREEN(in[idx])),
    (BLUE(in[idx_ref]) - BLUE(in[idx]))
  };

  int diff = diffs[0];
  if ((diffs[1] < 0 ? diffs[1] * -1 : diffs[1]) > diff) { diff = diffs[1]; }
  if ((diffs[2] < 0 ? diffs[2] * -1 : diffs[2]) > diff) { diff = diffs[2]; }

  int gray = 128 + diff;
  if (gray > 255) { gray = 255; }
  if (gray < 0) { gray = 0; }

  out[idx] = RGBA(gray, gray, gray, ALPHA(in[idx]));
}


#pragma mark Helper Methods

// Terminate program with message
void terminate(const char *fmt, ...) {
  va_list args;

  va_start(args, fmt);
  vfprintf(stderr, fmt, args);
  va_end(args);

  exit(1);
}

// Allocate a buffer large enough to store pixel data for given image.
uint32_t *alloc_image_buffer(Image *img) {
  return (uint32_t *) malloc(img->number_of_pixels * sizeof(uint32_t));
}

// Read an image from a file
static Image *read_image(const char *filename) {
  Image *img = (Image *) malloc(sizeof(Image));

  if (png_open_file_read(&img->png, filename) != PNG_NO_ERROR) {
    terminate("Couldn't open image\n");
  }

  // Number of pixels
  img->width            = img->png.width;
  img->height           = img->png.height;
  img->number_of_pixels = img->png.width * img->png.height;

  if (img->png.color_type != PNG_TRUECOLOR_ALPHA) {
    terminate("Only true color alpha images supported\n");
  }

  img->pixels = alloc_image_buffer(img);

  if (png_get_data(&img->png, (unsigned char *) img->pixels) != PNG_NO_ERROR) {
    terminate("Could not read image data\n");
  }

  return img;
}

// Save a transformed image.
static void save_image(const char *filename, uint32_t *img_data, const Image *orig_img) {
  png_t out;

  if (png_open_file_write(&out, filename) != PNG_NO_ERROR) {
    terminate("couldn't open image to save\n");
  }

  if (png_set_data(&out, orig_img->png.width, orig_img->png.height, orig_img->png.depth,
                   orig_img->png.color_type, (unsigned char *)img_data) != PNG_NO_ERROR) {
    terminate("Could not save image data\n");
  }

  png_close_file(&out);
}


#pragma mark Info Outputs

// Output CUDA information
static void showCudaInfo() {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  printf("CUDA INFORMATION\n================\n");
  printf("Name: %s\n", prop.name);
  printf("Total Memory: %u Bytes\n", prop.totalGlobalMem);
  printf("Max. Threads Per Block: %d\n", prop.maxThreadsPerBlock);
  printf("Clock Rate: %d kHz\n", prop.clockRate);
  printf("Multiprocessors: %d\n", prop.multiProcessorCount);
  printf("Concurrent Kernels: %d\n", prop.concurrentKernels);
}

// Output image information
static void showImageInfo(const Image *img) {
  printf("\nIMAGE INFORMATION\n================\n");
  printf("Width: %u\n", img->width);
  printf("Height: %u\n", img->height);
  printf("Total Pixels: %u\n", img->number_of_pixels);
}


#pragma mark main

int main(int argc, char **argv) {
  if (argc != 4) {
    terminate("Usage: %s <swap|gray|blur|emboss> <infile> <outfile>\n", __progname);
  }

  // Load image
  png_init(0, 0);

  Image    *img          = read_image(argv[2]);
  uint32_t *img_data     = img->pixels;
  uint32_t *out_img_data = (uint32_t *) alloc_image_buffer(img);

  // Show CUDA infos
  showCudaInfo();

  // Show image infos
  showImageInfo(img);

  // Initialize/allocate buffers
  size_t buffer_size = img->number_of_pixels * sizeof(uint32_t);
  uint32_t *dev_imgdata, *dev_imgdata_out;
  hipMalloc((void **) &dev_imgdata, buffer_size);
  hipMalloc((void **) &dev_imgdata_out, buffer_size);

  // Copy image data to device
  hipMemcpy(dev_imgdata, img_data, buffer_size, hipMemcpyHostToDevice);
  dim3 grid(img->width, img->height);

  // Switch transformation type
  if (strcmp(argv[1], "swap") == 0) {
    kernel_swap_green_blue<<<grid, 1>>>(dev_imgdata, dev_imgdata_out, img->width, img->height);
  } else if (strcmp(argv[1], "gray") == 0) {
    kernel_gray<<<grid, 1>>>(dev_imgdata, dev_imgdata_out, img->width, img->height);
  } else if (strcmp(argv[1], "blur") == 0) {
    kernel_blur<<<grid, 1>>>(dev_imgdata, dev_imgdata_out, img->width, img->height);
  } else if (strcmp(argv[1], "emboss") == 0) {
    kernel_emboss<<<grid, 1>>>(dev_imgdata, dev_imgdata_out, img->width, img->height);
  } else {
    terminate("\nUnsupported Transformation: %s\n", argv[1]);
  }

  // Copy transformed image data from device
  hipMemcpy(out_img_data, dev_imgdata_out, buffer_size, hipMemcpyDeviceToHost);

  // Save image to disk and close file handle
  save_image(argv[3], out_img_data, img);
  png_close_file(&img->png);

  printf("\nSaved Transformed Image: %s\n", argv[3]);

  // Cleanup memory
  free(img->pixels);
  free(img);
  free(out_img_data);

  return 0;
}
